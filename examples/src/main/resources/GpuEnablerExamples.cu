
#include <hip/hip_runtime.h>

extern "C"
// another simple test kernel
__global__ void multiplyBy2(int *size, int *in, int *out) {
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < *size) {
        out[ix] = in[ix] * 2;
    }
}


extern "C"
// test reduce kernel that sums elements
__global__ void sum(int *size, int *input, int *output, int *stage, int *totalStages) {
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    const int jump = 64 * 256;
    // if (ix == 0) printf("size: %d stage : %d totalStages : %d \n",*size, *stage, *totalStages);
    if (*stage == 0) {
        if (ix < *size) {
            assert(jump == blockDim.x * gridDim.x);
            int result = 0;
            for (long i = ix; i < *size; i += jump) {
                result += input[i];
            }
            input[ix] = result;
        }
    } else if (ix == 0) {
        const long count = (*size < (long)jump) ? *size : (long)jump;
        int result = 0;
        for (long i = 0; i < count; ++i) {
            result += input[i];
        }
        output[0] = result;
    }
}


extern "C"
// test reduce kernel that sums elements
__global__ void sum1(int *size, int *input, int *output) {
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    if (ix == 0) {
        int result = 0;
        for (long i = ix; i < *size; i++) {
            result += input[i];
        }
        output[0] = result;
    }
}


extern "C"
__global__ void add(int n, long *a, long *b, long *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        sum[i] = a[i] + b[i];
        printf("CUDA KERNEL ADD %ld + %ld = %ld \n",a[i],b[i],sum[i]);
    }

}

extern "C"
__global__ void mul(int n, long *a, long *b, long *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        sum[i] = a[i] * b[i];
        printf("CUDA KERNEL MUL %ld * %ld = %ld \n",a[i],b[i],sum[i]);
    }

}

extern "C"
__global__ void arrayTest(int n, long *factor, long *arr,int *const_arr1, long *const_arr2,long *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i == 0) {
       printf("In ArrayTest n=%d factor=%p arr=%p result=%p \n",n,factor,arr,result);
       printf("In const %d %d %d\n",const_arr1[0],const_arr1[1],const_arr1[2]);
       printf("In const %ld %ld %ld\n",const_arr2[0],const_arr2[1],const_arr2[2]);
    }

    if (i<n)
    {
        int idx = i * 3;
        result[idx]=arr[idx] * factor[i];
        result[idx + 1]=arr[idx + 1] * factor[i];
        result[idx + 2]=arr[idx + 2] * factor[i];
        printf("ArrayTest  [%ld] * [%ld %ld %ld] = [%ld %ld %ld] \n", factor[i],
                 arr[idx],arr[idx+1],arr[idx+2],
                result[idx],result[idx+1],result[idx+2]);
    }

}

extern "C"
__global__ void arrayMult(int n, long *factor, long *arr,int *const_arr1, long *const_arr2,long *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i == 0) {
       printf("In ArrayTest n=%d factor=%p arr=%p result=%p \n",n,factor,arr,result);
       printf("In const %d %d %d\n",const_arr1[0],const_arr1[1],const_arr1[2]);
       printf("In const %ld %ld %ld\n",const_arr2[0],const_arr2[1],const_arr2[2]);
    }

    if (i<n)
    {
        int idx = i * 3;
        result[idx]=arr[idx] * factor[i];
        result[idx + 1]=arr[idx + 1] * factor[i];
        result[idx + 2]=arr[idx + 2] * factor[i];
        printf("ArrayTest  [%ld] * [%ld %ld %ld] = [%ld %ld %ld] \n", factor[i],
                 arr[idx],arr[idx+1],arr[idx+2],
                result[idx],result[idx+1],result[idx+2]);
    }

}


extern "C"
__global__ void arrayTestMod(int n, long *factor, long *arr, long *result, int *const_arr1, long *const_arr2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i == 0) {
       printf("In ArrayTest n=%d factor=%p arr=%p result=%p \n",n,factor,arr,result);
       printf("In const %d %d %d\n",const_arr1[0],const_arr1[1],const_arr1[2]);
       printf("In const %ld %ld %ld\n",const_arr2[0],const_arr2[1],const_arr2[2]);
       int idx = i * 3;
       result[idx]=arr[idx] * factor[i];
       printf("ArrayTest  [%ld] * [%ld %ld %ld] = [%ld] \n", factor[i],
          arr[idx],arr[idx+1],arr[idx+2],
                result[idx]);
    }
}

extern "C"
__global__ void arrayTestModStages(int n, long *factor, long *arr, long *result, int *const_arr1, long *const_arr2, 
     long *const_fixed, int outputArraySize, int stage, int totalstages)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i == 0) {
       printf("N : %d, stage: %d, totalstages: %d \n", n, stage, totalstages); 
       printf("In outputArraySize : %d \n", outputArraySize);
       printf("In ArrayTest n=%d factor=%p arr=%p result=%p \n",n,factor,arr,result);
       printf("In Fixed const %ld .. %ld \n",const_fixed[0], const_fixed[34]);
       printf("In const %d %d %d\n",const_arr1[0],const_arr1[1],const_arr1[2]);
       printf("In const %ld %ld %ld\n",const_arr2[0],const_arr2[1],const_arr2[2]);
       int idx = i * 3;
       result[idx]=arr[idx] * factor[i];
       printf("ArrayTest  [%ld] * [%ld %ld %ld] = [%ld] \n", factor[i],
          arr[idx],arr[idx+1],arr[idx+2],
                result[idx]);
    }
}



extern "C"
// another simple test kernel
__global__ void multiplyBy2_self(int size, long *inout) {
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < size) {
        inout[ix] = inout[ix] * 2;
    }
}
